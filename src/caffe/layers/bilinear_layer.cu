#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/vision_layers.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

__global__ void sync_patches() { }

template <typename Dtype>
void BilinearLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  for (int input_idx = 0; input_idx < 2; ++input_idx) {
    // First, we transform input tensors using im2col in order to localize
    // patches.
    im2col_temp_bottom_vec_[0] = bottom[input_idx];
    im2col_temp_top_vec_[0] = im2col_top_vec_[input_idx];
    im2col_layers_[input_idx]->Forward(im2col_temp_bottom_vec_, 
                                       im2col_temp_top_vec_);

    // Next, we transpose resulting tensors so that patch index dimension 
    // becomes the slowest changing dimension.
    int num_rows = im2col_top_vec_[input_idx]->count() / num_patches_per_image_;
    int num_cols = num_patches_per_image_;
  
    const Dtype* im2col_top_data = im2col_top_vec_[input_idx]->gpu_data();
    Dtype* geam_top_data = geam_top_vec_[input_idx]->mutable_gpu_data();

    caffe_gpu_geam<Dtype>(CblasTrans, CblasTrans, num_cols, num_rows,
        (Dtype) 1., im2col_top_data, im2col_top_data, (Dtype) 0.,
        geam_top_data);
  }

  // We use GEMM to get the bilinear maps for all the patches.
  // Total number of patches is batch_size x num_patches_per_image.
  int num_patches = num_ * num_patches_per_image_;
  int M = channels_a_;
  int N = channels_b_;
  int K = kernel_count_;

  for (int patch_index = 0; patch_index < num_patches; ++patch_index) {
    const Dtype* patch_a_data = geam_top_vec_[0]->gpu_data();
    const Dtype* patch_b_data = geam_top_vec_[1]->gpu_data();
    Dtype* gemm_top_data = gemm_top_.mutable_gpu_data();

    patch_a_data += patch_index * M * K;
    patch_b_data += patch_index * N * K;
    gemm_top_data += patch_index * M * N;

    Caffe::set_cublas_stream(streams_[patch_index % streams_.size()]);

    caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasTrans, M, N, K, (Dtype) 1.,
                          patch_a_data, patch_b_data, (Dtype) 0., 
                          gemm_top_data);
  }
  // Synchronize the work across patches, each of which went into its own
  // stream, by launching an empty kernel into the default (null) stream.
  // NOLINT_NEXT_LINE(whitespace/operators)
  sync_patches<<<1, 1>>>();

  Caffe::set_cublas_stream(0);

  // Finally, we transpose patch index dimension back.
  int num_rows = num_patches_per_image_;
  int num_cols = top[0]->count() / num_patches_per_image_;

  const Dtype* gemm_top_data = gemm_top_.gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();

  caffe_gpu_geam<Dtype>(CblasTrans, CblasTrans, num_cols, num_rows,
      (Dtype) 1., gemm_top_data, gemm_top_data, (Dtype) 0.,
      top_data);
}

template <typename Dtype>
void BilinearLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  // During backpropagation we reverse the order of operations.

  // (Un)transpose top diffs.
  int num_rows = top[0]->count() / num_patches_per_image_;
  int num_cols = num_patches_per_image_;

  const Dtype* top_diff = top[0]->gpu_diff();
  Dtype* gemm_top_diff = gemm_top_.mutable_gpu_diff();

  caffe_gpu_geam<Dtype>(CblasTrans, CblasTrans, num_cols, num_rows,
      (Dtype) 1., top_diff, top_diff, (Dtype) 0.,
      gemm_top_diff);

  // GEMM gradients.
  int num_patches = num_ * num_patches_per_image_;
  int M = channels_a_;
  int N = channels_b_;
  int K = kernel_count_;

  // Gradient with respect to the left GEMM operand.
  if (propagate_down[0]) {
    for (int patch_index = 0; patch_index < num_patches; ++patch_index) {
      const Dtype* patch_b_data = geam_top_vec_[1]->gpu_data();
      const Dtype* gemm_top_diff = gemm_top_.gpu_diff();
      Dtype* patch_a_diff = geam_top_vec_[0]->mutable_gpu_diff();

      patch_b_data += patch_index * N * K;
      gemm_top_diff += patch_index * M * N;
      patch_a_diff += patch_index * M * K;

      Caffe::set_cublas_stream(streams_[patch_index % streams_.size()]);

      caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, M, K, N, (Dtype) 1.,
                            gemm_top_diff, patch_b_data, (Dtype) 0.,
                            patch_a_diff);
    }
  }

  // Gradient with respect to the right GEMM operand.
  if (propagate_down[1]) {
    for (int patch_index = 0; patch_index < num_patches; ++patch_index) {
      const Dtype* patch_a_data = geam_top_vec_[0]->gpu_data();
      const Dtype* gemm_top_diff = gemm_top_.gpu_diff();
      Dtype* patch_b_diff = geam_top_vec_[1]->mutable_gpu_diff();

      patch_a_data += patch_index * M * K;
      gemm_top_diff += patch_index * M * N;
      patch_b_diff += patch_index * N * K;

      Caffe::set_cublas_stream(streams_[patch_index % streams_.size()]);

      caffe_gpu_gemm<Dtype>(CblasTrans, CblasNoTrans, N, K, M, (Dtype) 1.,
                            gemm_top_diff, patch_a_data, (Dtype) 0.,
                            patch_b_diff);
    }
  }
  // Synchronize the work across patches, each of which went into its own
  // stream, by launching an empty kernel into the default (null) stream.
  // NOLINT_NEXT_LINE(whitespace/operators)
  sync_patches<<<1, 1>>>();

  // Gradients with respect to bottom data.
  for (int input_idx = 0; input_idx < 2; ++input_idx) {
    if (!propagate_down[input_idx]) {
      continue;
    }
    
    // Again, we (un)transpose diffs so that they are suitable for the
    // col2im op.
    int num_rows = num_patches_per_image_;
    int num_cols = im2col_top_vec_[input_idx]->count() / num_patches_per_image_;
  
    const Dtype* geam_top_diff = geam_top_vec_[input_idx]->gpu_diff();
    Dtype* im2col_top_diff = im2col_top_vec_[input_idx]->mutable_gpu_diff();

    caffe_gpu_geam<Dtype>(CblasTrans, CblasTrans, num_cols, num_rows,
        (Dtype) 1., geam_top_diff, geam_top_diff, (Dtype) 0.,
        im2col_top_diff);

    // Finally, we invoke bprop routine of the internal im2col layer.
    im2col_temp_top_vec_[0] = im2col_top_vec_[input_idx];
    im2col_propagate_down_[0] = propagate_down[input_idx];
    im2col_temp_bottom_vec_[0] = bottom[input_idx];
    im2col_layers_[input_idx]->Backward(im2col_temp_top_vec_,
                                        im2col_propagate_down_,
                                        im2col_temp_bottom_vec_);
  }

}

INSTANTIATE_LAYER_GPU_FUNCS(BilinearLayer);

}  // namespace caffe
